
#include <hip/hip_runtime.h>
#include <complex>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hip/hip_complex.h>

// Kernel Definitions

/******************************************************************************
 * Function: CUDAisInMandelbrotSet
 *
 * Authors: Elliott Rarden & Katie Macmillan
 *
 * Description: 
 * CUDAisInMandelbrotSet() takes in cuDoubleComplex number, c, and an int pointer
 * to return the number of iterations
 *
 * Parameters:		[in]	c
 *				[in-out]	iterations
 ******************************************************************************/
__device__ void CUDAisInMandelbrotSet(hipDoubleComplex c, int *iterations){
	double zr = 0;
	double zi = 0;
	hipDoubleComplex z = make_hipDoubleComplex(zr, zi);
	int i = 0;

	for (i = 0; i < 100 && hipCabs(z) < 2.0; ++i){
		z = hipCadd(hipCmul(z, z), c); // z = z*z + c;
	}

	*iterations = i;
}

/******************************************************************************
 * Function: CUDAisInMandelbrotSet
 *
 * Authors: Elliott Rarden & Katie Macmillan
 *
 * Description: 
 * CUDAisInJuliaSet() takes in two cuDoubleComplex numbers, c & z, and an int
 * pointer to return the number of iterations
 *
 * Parameters:		[in]	z
 *					[in]	c
 *				[in-out]	iterations
 ******************************************************************************/
__device__ void CUDAisInJuliaSet(hipDoubleComplex z, hipDoubleComplex c, int *iterations){
	int i = 0;

	for (i = 0; i < 100 && hipCabs(z) < 2.0; ++i){
		z = hipCadd(hipCmul(z, z), c); // z = z*z + c;
	}

	*iterations = i;
}

/******************************************************************************
 * Function: mandelCalc
 *
 * Authors: Elliott Rarden & Katie Macmillan
 *
 * Description: 
 * mandelCalc() is a kernel which takes in a set of real and imaginary values
 * and will calcualte in parallel wether or not each point is in or out of the
 * mandelbrot set.
 *
 * Parameters:		[in]	cReals
 *					[in]	cImags
 *				[in-out]	iterations
 *					[in]	len
 ******************************************************************************/
__global__ void mandelCalc(double *cReals, double *cImags, int *iterations, int len) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	hipDoubleComplex num;

	if(i < len){
		num = make_hipDoubleComplex(cReals[i], cImags[i]);
		CUDAisInMandelbrotSet(num, &iterations[i]);
	}
}

/******************************************************************************
 * Function: juliaCalc
 *
 * Authors: Elliott Rarden & Katie Macmillan
 *
 * Description: 
 * juliaCalc() is a kernel which takes in a set of real and imaginary values as
 * well as a real and imaginary number (as doubles), and will calcualte in
 * parallel wether or not each point in the arrays is in or out of the Julia set
 * associated with the given point
 *
 * Parameters:		[in]	zReal
 *					[in]	zImag
 *					[in]	cReals
 *					[in]	cImags
 *				[in-out]	iterations
 *					[in]	len
 ******************************************************************************/
__global__ void juliaCalc(double zReal, double zImag, double *cReals, double *cImags, int *iterations, int len) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	hipDoubleComplex num;
	hipDoubleComplex z;

	if(i < len){
		num = make_hipDoubleComplex(cReals[i], cImags[i]);
		z = make_hipDoubleComplex(zReal, zImag);
		CUDAisInJuliaSet(num, z, &iterations[i]);
	}
}

/******************************************************************************
 * Function: cudaCalcMandelbrot
 *
 * Authors: Elliott Rarden & Katie Macmillan
 *
 * Description: 
 * cudaCalcMandelbrot() is a function which takes in a set of real and imaginary
 * values, and will reutrn if each point is in our out of the mandelbrot set using
 * the in-out iterations parameter
 *
 * Parameters:		[in]	setOfReals
 *					[in]	setOfImags
 *				[in-out]	iterations
 *					[in]	len
 ******************************************************************************/
__host__ void cudaCalcMandelbrot(double *setOfReals, double *setOfImags, int *iterations, int len) {
	// Block management
	int n = len;

	// Device arrays
	double *d_set_reals;
	double *d_set_imags;
	int *d_iterations;

	// Allocate our memory on the device
	hipMalloc(&d_set_reals,	len * sizeof(double));
	hipMalloc(&d_set_imags,	len * sizeof(double));
	hipMalloc(&d_iterations,	len * sizeof(int)); 

	// Copy our stuff to the device
	hipMemcpy(d_set_reals,		setOfReals,	len * sizeof(double),	hipMemcpyHostToDevice);
	hipMemcpy(d_set_imags,		setOfImags,	len * sizeof(double),	hipMemcpyHostToDevice);
	hipMemcpy(d_iterations,	iterations,	len * sizeof(int),	hipMemcpyHostToDevice);

	// Run the code on the GPU
	int nThreads = 128; // should be multiple of 32 (up to 1024)
	int nBlocks = ( n + nThreads - 1 ) / nThreads;
	mandelCalc<<<nBlocks, nThreads>>>(d_set_reals, d_set_imags, d_iterations, len);

	// Copy stuff from the GPU to our host
	hipMemcpy(setOfReals,	d_set_reals,	len * sizeof(double),	hipMemcpyDeviceToHost);
	hipMemcpy(setOfImags,	d_set_imags,	len * sizeof(double),	hipMemcpyDeviceToHost);
	hipMemcpy(iterations,	d_iterations,	len * sizeof(int),	hipMemcpyDeviceToHost);

	// Free the device memory
	hipFree(d_set_reals);
	hipFree(d_set_imags);
	hipFree(d_iterations);
}

/******************************************************************************
 * Function: cudaCalcJulia
 *
 * Authors: Elliott Rarden & Katie Macmillan
 *
 * Description: 
 * cudaCalcJulia() is a function which takes in a real and imaginary z value and
 * a set of real and imaginary values, c, and will reutrn if each point in c is
 * in the Julia set associated with the point z
 *
 * Parameters:		[in]	zReal
 *					[in]	zImag
 *					[in]	cReals
 *					[in]	cImags
 *				[in-out]	iterations
 *					[in]	len
 ******************************************************************************/
__host__ void cudaCalcJulia(double zReal, double zImag, double *setOfReals, double *setOfImags, int *iterations, int len) {
	// Local variables
	int n = len;

	// Device variables
	double *d_set_reals;
	double *d_set_imags;
	int *d_iterations;

	// Allocate our memory on the device
	hipMalloc(&d_set_reals,	len * sizeof(double));
	hipMalloc(&d_set_imags,	len * sizeof(double));
	hipMalloc(&d_iterations,	len * sizeof(int));

	// Copy our stuff to the device
	hipMemcpy(d_set_reals,		setOfReals,		len * sizeof(double),			hipMemcpyHostToDevice);
	hipMemcpy(d_set_imags,		setOfImags,		len * sizeof(double),			hipMemcpyHostToDevice);
	hipMemcpy(d_iterations,	iterations,		len * sizeof(int),				hipMemcpyHostToDevice);

	// Run the code on the GPU
	int nThreads = 128; // should be multiple of 32 (up to 1024)
	int nBlocks = ( n + nThreads - 1 ) / nThreads;
	juliaCalc<<<nBlocks, nThreads>>>(zReal, zImag, d_set_reals, d_set_imags, d_iterations, len);

	// Copy stuff from the GPU to our host
	hipMemcpy(iterations,	d_iterations,	len * sizeof(int),	hipMemcpyDeviceToHost);

	// Free the device memory
	hipFree(d_set_reals);
	hipFree(d_set_imags);
	hipFree(d_iterations);
}
